#include "hip/hip_runtime.h"
/************************************************************
This code solves the NP hard problem of QUBO (Quadratic unconstrained Binary Optimization) 
It follows the Adaptive Bulk Search Algorithm described in the paper 
Ryota Yasudo, Koji Nakano, Yasuaki Ito, Masaru Tatekawa, Ryota Katsuki, Takashi Yazane, and Yoko Inaba. 2020. 
Adaptive Bulk Search: Solving Quadratic Unconstrained Binary Optimization Problems on Multiple GPUs. 
In 49th International Conference on Parallel Processing - ICPP (ICPP '20). Association for Computing Machinery, New York, NY, USA, Article 62, 1–11.
 https://doi.org/10.1145/3404397.3404423
This is implemented in CUDA C++
to compile in linux use ./compile.sh after changing necessary compile flags
the options to be provided are
-i filename #compulsory options filename containing the qubo matrix in qubo format
#optional options include
-it  no of iterations ofr local search
-nsol no of parallel instances of solutions to generate

**********************************************************/
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <vector> 
#include <algorithm>
#include <utility>
#include <random>
#include <iterator>
#include <iostream>
#define the CUDA_API_PER_THREAD_DEFAULT_STREAM //enable concurrency
#include <hip/hip_runtime.h>
#include "host.cuh"

using namespace std;
#define BITS_PER_THREADS 4

///combined search function straight search+ local search
__global__ void search(float *qubo,int nNodes,int nsol,bool *d_tbuffer,bool *d_sbuffer,float *d_epool,int iterations,int *d_count,int *d_tcount);

int main(int argc, char *argv[])
{
  char *inFileName = NULL;
  FILE *inFile = NULL;
  //srand(time(NULL));
  srand(10);
  cout << "You have entered " << argc
       << " arguments:" << "\n";
  int niterations=1000;
  int nsol=4; //no of solutions in solution buffers
  int nmutations; //no of mutations
  for(int i=1;i<argc;++i)
    {
      cout << argv[i] << "\n";
      if(strstr(argv[i], "-i"))  //argv[i]=="-"&&argv[i][1]=="i")
	{
	  inFileName=argv[i+1];
	  break;
	}
      if(strstr(argv[i], "-it"))  //argv[i]=="-"&&argv[i][1]=="i")
	{
	  niterations=atoi(argv[i+1]);
	  break;
	}
      if(strstr(argv[i], "-nsol"))  //argv[i]=="-"&&argv[i][1]=="i")
	{
	  nsol=atoi(argv[i+1]);
	  break;
	}
    }
  
  printf("supplied file name is %s\n",inFileName);
  cout<<"iterations = "<<niterations<<endl;
  //read file
  inFile = fopen(inFileName, "r");
  
  int nmin,nmax,nNodes;
  read_qubo(inFile,nmin,nmax,nNodes);
  
  cout<<"found nodes "<<nNodes<<"("<<nmin<<","<<nmax<<")"<<endl;
  //fill qubo matrix
  float **val;
  val= new float *[nNodes];
  for(int i = 0; i <nNodes; i++) val[i] = new float[nNodes];
  fill_qubo(inFile,val,nmin);
  fclose(inFile);
  
  //parmateres
  
  nmutations=nNodes/4;
  //make qubo matrix lower triangular
  //	LowerTriangulize(val,nNodes);
  //initialization
  //solution pool
  bool *h_solpool=new bool [nsol*nNodes];
  float *h_epool=new float [nsol];
  //generate random initial solution buffer and calclaute their energies in h_epool
  initSol(h_solpool,nNodes,h_epool,nsol,val);

    //index vector
  int *h_indsol;
  h_indsol=new int [nsol];
  
  //sort solutions and get indices 
  sortSolution(h_epool,nsol,h_indsol);
  
  cout<<" energy after sorting from pool "<<endl;
  for(int i=0;i<nsol;i++)
    {
      cout<<h_epool[i]<<" "<<evaluate(h_solpool+h_indsol[i]*nNodes,val,nNodes)<<endl;
      for(int j=0;j<nNodes;j++)
	cout<<h_solpool[h_indsol[i]*nNodes+j];
      cout<<endl;
    }
  
  hipError_t err;
  const size_t malloc_limit = size_t(1024) * size_t(1024) * size_t(1024);
  hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit); 
  //allocate device memory
  ////device target buffer
  size_t size = nNodes *nsol* sizeof(bool); 
//target buffer
  bool *h_tbuffer=0;
  err=hipHostMalloc((void **)&h_tbuffer, size,hipHostMallocWriteCombined);
  if(err!=hipSuccess ) cout<<"unable to allocate host target buffer"<<hipGetErrorString(err)<<endl;
  
  
  //solution energy buffer
  float *h_ebuffer;
  err=hipHostMalloc((void **)&h_ebuffer, nsol*sizeof(float));
  if(err!=hipSuccess ) cout<<"unable to allocate host solution buffer"<<hipGetErrorString(err)<<endl;
  
  
  //copy initial solution to target buffer
  for(int k=0;k<nsol*nNodes;k++)
    {
      h_tbuffer[k]=h_solpool[k];
    }
  
  //allocate 1 dimensional array for qubo to facilitate transfer
  float *qubo;
  qubo=new float [nNodes*nNodes];
  int id;
  for(int i=0;i<nNodes;i++)
    {
      for(int j=0;j<nNodes;j++)
	{
	  id=i*nNodes+j;
	  qubo[id]=val[i][j];
	}
    }
  
  
  bool *d_tbuffer=NULL;
  err=hipMalloc((void **)&d_tbuffer, size);
  if(err!=hipSuccess ) cout<<"unable to allocate target buffer"<<hipGetErrorString(err)<<endl;
  
  ////device best solution buffer
  bool *d_sbuffer=NULL;
  err=hipMalloc((void **)&d_sbuffer, size);
  if(err!=hipSuccess ) cout<<"unable to allocate device solution buffer"<<hipGetErrorString(err)<<endl;
  ////host best solution buffer to hold solutions and insert in the solution pool
  bool *h_sbuffer=0;
  err=hipHostMalloc((void **)&h_sbuffer, size);
  if(err!=hipSuccess ) cout<<"unable to allocate host solution buffer"<<hipGetErrorString(err)<<endl;
  
  // the device qubo matrix
  size_t size2d=nNodes*nNodes*sizeof(float);
  float *d_qubo = NULL;
  err=hipMalloc((void **)&d_qubo, size2d);
  if(err!=hipSuccess ) cout<<"unable to allocate qubo"<<hipGetErrorString(err)<<endl;

  //copy from host to device
  err=hipMemcpy(d_qubo, qubo, size2d, hipMemcpyHostToDevice);
  if(err!=hipSuccess ) cout<<"qubo copy failed "<<hipGetErrorString(err)<<endl;
  
  
  err=hipMemcpy(d_tbuffer, h_tbuffer, size, hipMemcpyHostToDevice);
  if(err!=hipSuccess ) cout<<"target buffer copy failed "<<hipGetErrorString(err)<<endl;
  
	    // the device energy pool
  size_t sizeE=nsol*sizeof(float);
  float *d_epool ;
  err=hipMalloc(&d_epool, sizeE);
  if(err!=hipSuccess ) cout<<"unable to allocate energy pool"<<hipGetErrorString(err)<<endl;
  /*****************************************************/
  //streams
  hipStream_t stream1,stream2;
  hipStreamCreate(&stream1); //d2h counter
  hipStreamCreate(&stream2);  //h2d counter
  //stream for d2h
  hipStream_t streamd2h,	streamh2d;
  hipStreamCreate(&streamd2h);
  hipStreamCreate(&streamh2d);
  // create cuda event handles
  hipEvent_t start, stop,copy;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&copy);
  //run the actual function localsearch
  /**************/
  //counters
  //solution buffer count
  int *h_count = 0;
  err=hipHostMalloc((void **)&h_count, nsol*sizeof(int));
    if(err!=hipSuccess ) cout<<"could not allocate h_count "<<endl;

  int *d_count = 0;
  err=hipMalloc((void **)&d_count, nsol*sizeof(int));
      if(err!=hipSuccess ) cout<<"could not allocate d_count "<<endl;

  //target buffer count
   int *h_tcount = 0;
  err=hipHostMalloc((void **)&h_tcount, nsol*sizeof(int),hipHostMallocWriteCombined);
        if(err!=hipSuccess ) cout<<"could not allocate h_tcount "<<endl;

  int *d_tcount = 0;
  err=hipMalloc((void **)&d_tcount, nsol*sizeof(int));
          if(err!=hipSuccess ) cout<<"could not allocate d_tcount "<<endl;

  //initialize
  int h_count_old[nsol];
  
  for(int i=0;i<nsol;i++)
    {
      h_count[i]=0;
      h_tcount[i]=1;
    }
  ////send to device
  hipEventRecord(start, 0);
  
  hipMemcpy(d_count,h_count,nsol*sizeof(int),hipMemcpyHostToDevice);	
  hipMemcpy(d_tcount,h_tcount,nsol*sizeof(int),hipMemcpyHostToDevice);	 		
  
  /*************************************************************/
  cout<<"cuda local search test"<<endl;
  static const int nthreads=nNodes/BITS_PER_THREADS;
  ///combined search function straight search+ local search
  search<<<nsol,nthreads,0,0>>>(d_qubo,nNodes,nsol,d_tbuffer,d_sbuffer,d_epool,niterations,d_count,d_tcount);
  cout<<"test exited "<<endl;
  
  err = hipGetLastError();
  
  if(err!=hipSuccess ) cout<<"LOCAL SEARCH failed "<<hipGetErrorString(err)<<endl;
  hipEventRecord(stop, 0);
  /****event **************/
  //create event for copy purpose
  hipEvent_t copys[nsol],copyt[nsol];
  for(int i=0;i<nsol;i++)
    {
      hipEventCreate(&copys[i]);
      hipEventCreate(&copyt[i]);
    }
  
  /****************************/
  //cpu code
  std::vector<int>rec;
  int counter=0;
  for(int k=0;k<nsol;k++) h_count_old[k]=h_count[k];
  while (hipEventQuery(stop) == hipErrorNotReady) 
    {
      //h_count++;
      
      hipMemcpyAsync(h_count,d_count,nsol*sizeof(int),hipMemcpyDeviceToHost,stream1);	
      hipEventRecord(copy, stream1);
      
      if(hipEventQuery(copy)==hipSuccess)
	{
	  int newsols=0; //no of new solutions generated
	  for(int k=0;k<nsol;k++)
	    {newsols+=h_count[k]-h_count_old[k];}
	  rec.push_back(newsols);
	  if(newsols>0)
	    {
	      counter++;
	      cout<<counter<<" new sols found ="<<newsols<<endl;
	      hipMemcpyAsync(h_sbuffer,d_sbuffer,nNodes*sizeof(bool),hipMemcpyDeviceToHost,stream1);
		  hipMemcpyAsync(h_ebuffer,d_epool,sizeof(float),hipMemcpyDeviceToHost,stream1);
		  hipEventRecord(copys[0], stream1);
	     
		int istart;
		//wait for completion of copy of solution from device
		do{} while(hipEventQuery(copys[0])!=hipSuccess);
	      //update solution pool
	      for(int k=0;k<nsol;k++)
		{
		  if((h_count[k]>h_count_old[k])&& (hipEventQuery(copys[0])==hipSuccess))
		    {
			  //update solution pool
			  istart=k*nNodes; 
			  insertSol(h_ebuffer[k],h_epool,nsol,h_indsol,h_sbuffer+istart,h_solpool,nNodes);
		      cout<<"new solutions inserted at "<<k<<endl;
		    }
		}
	      //generate new solutions
	      
	      for(int k=0;k<nsol;k++)
			{
			if(h_count[k]>h_count_old[k])
				{
				GAreprod(h_solpool,nsol,h_tbuffer+k*nNodes,nNodes,nmutations);
				cout<<"new sol generated for "<<k<<" "<<endl;
				}
			}
	      cout<<endl;
	      ///copy the new solutions to the device
	      for(int k=0;k<nsol;k++)
		{
		  if(h_count[k]>h_count_old[k])
		    {
		      istart=k*nNodes; 
		      hipMemcpyAsync(d_tbuffer+istart,h_tbuffer+istart,nNodes*sizeof(bool),hipMemcpyHostToDevice,stream2);
		      	      
		      h_tcount[k]++;
			cout<<"new sols sent for k ="<<k<<endl; 
			  //update the counter
			h_count_old[k]=h_count[k];

		      err = hipGetLastError();
		      
		      if(err!=hipSuccess ) cout<<"error  "<<hipGetErrorString(err)<<endl;
		      
		      

		    }
		}
		for(int k=0;k<nsol;k++)
			cout<<" k= "<<k<<" count="<<h_tcount[k]<<endl;
				      //send counter also
		  hipEvent_t ccount;
		  hipEventCreate(&ccount);
			  hipMemcpyAsync(d_tcount,h_tcount,nsol*sizeof(int),hipMemcpyHostToDevice,stream2);
			  hipEventRecord(ccount, stream2);
	    
	      err = hipGetLastError();
	      
	      if(err!=hipSuccess ) cout<<"copy failed "<<hipGetErrorString(err)<<endl;
	      	do{}
			while(hipEventQuery(ccount)!=hipSuccess);
	      cout<<"copied data and counts to device successfully "<<endl;
	    }
	  //send the cpounter to device
	  
	}
    }
  
  //complete updating target buffer
  hipStreamSynchronize(streamh2d);
  hipStreamSynchronize(streamd2h);
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);
  //finalize
  hipDeviceSynchronize();
  
  //print the best solution
  cout<<"best solution found so far "<<h_epool[0]<<endl;
  int k=h_indsol[0];
  for(int i=0;i<nNodes;i++)
    {
      cout<<h_solpool[k*nNodes+i];
    }
  cout<<endl;
  cout<<" energy check: " <<endl;
  for(int i=0;i<nsol;i++)
    {
      k=h_indsol[i];
      cout<<h_epool[i]<<"  "<<k<<" "<<evaluate(h_solpool+k*nNodes,val,nNodes)<<endl;
    }
  
  printf("CPU generated new solution %d times and updated to target buffer\n", counter);
  hipDeviceSynchronize();
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
   
  /********************************************/
  
  //free streams
  hipStreamDestroy(stream1);
  hipStreamDestroy(streamd2h);
  hipStreamDestroy(streamh2d);
  // Free device memory
  hipFree(d_qubo);
  hipFree(d_tbuffer);
  hipFree(d_sbuffer);
  hipFree(d_epool);
  hipFree(d_count);
  hipFree(d_tcount);
  //free host memory 
  hipHostFree(h_ebuffer);
  hipHostFree(h_sbuffer);
  hipHostFree(h_tbuffer);
  hipHostFree(h_count);
  hipHostFree(h_tcount);
  delete [] qubo;
  for(int i=0;i<nNodes;i++)
    delete [] val[i];
  delete [] val;
  delete h_solpool;
  delete h_epool;
  delete h_indsol;
  hipDeviceReset();
  
  return 0;
}
